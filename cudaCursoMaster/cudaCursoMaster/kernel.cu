#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>

/*
------------------------102- Introducction to parallel programing -------------------------------------------------------------------------------------------------------------------------------------------------------

Context (instruccion por turnos)
	- Collection of data about process which allows processor to suspend or hold the execution of a process and restart the execution later.
	- Memory addresses
	- Program counter states

Thread (secuendia mas peque�a de instruccion programada
	- Process
		- threads (subprocess)

Parallel Process
	- Tipos
		- Paralelismo a nivel de tarea
			- los nucleos realizan tareas distintas con datos distintos o los mismos
		- Paralelismo a nivel de datos
			- los nucleos realizan la misma tarea con diferentes datos

Paralelismo vs concurrencia.
	- Concurrencia = realizacion de procesos en distintos tiempos(secuenciales) de milesimas de segundo que aparentan simultaneidad o paralelismo
	- Paralelismo = distintos nucleos realizan tareas al mismo tiempo

------------------------104- Install -------------------------------------------------------------------------------------------------------------------------------------------------------
Revisar compatibilidad en wikipedia en ingles.
GPGPU- 
windows + r (dxdiag) // visualiza las caracteristicas del PC 
windows + r (cmd) //escribir (nvcc --version) para saber la version de CUDA instalado del PC

*/

//------------------------105 - Basic steps of a CUDA program----------------------------------------------------
/*RESUMEN
* - initization of data from CPU
* - transfer data from CPU context to GPU context
* - Kernel launc with needed grid/block size
* - Transfer results back to CPU context from CPU context
* - Reclaim the memory from both CPU and GPU
* 
* - IMPORTANTE
*	-Grid - Grid is a collection of all the threads launch for a kernel ( coleccion de todos los hilos lanzados para un kernel)
*		- En el ejemplo hello CUDA world se tienen 20 subprocesos, los hilos en una cuadricula estan organizados
*		  en un grupo llamado bloques de hilos
*	-Block - subconjunto de hilos dentro de un GRID que se pueden representar como un cubo(3d) mas peque�o que a su vez esta subdividido en peque�os cubos que representan a los hilos o threads
	
	-GRID (cubo general en x, y z)
		-BLOCK (subcubo dentro de GRID que forma un subconjunto de hilos)
			-THREADS

	kernel_ name <<<
				number_of_blocks, // especifica cuantos bloques de hilos en la cuadracula en cada dimension
				thread_per_block  // especifica cuantos hilos en un bloque en cada dimension
								>>> (arguments) // TODO ESTO ES EN UNA DIMENSION

* - Para especificar cuadriculas y bloques multidimensionales
*	-dim3 variable_name (x,y,z) // se inicializa por defecto en 1
*		- dim3 variable_name(x,y,z) // puede acceder a cada valor de dimension 
*			- variable_name.x
* *			- variable_name.y
* *			- variable_name.z

*EJEMPLO UNIDIMENSIONAL
* - 8 bloques de hilos, donde cada bloque tiene 4 hilos en la dimension x
	-la dimension de nuestro bloque es de cuatro hilos en la dimension x y 1 hilo en las dimensiones Y y Z.
		
		-dim3 grid(8,1,1) // nos referimos a todos los hilos lanzados para un kernel como grid.
		-dim3 block(4,1,1)
		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________
		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |
		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |	
		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|

 8 bloques unidimensionales con 8 hilos unidimensionales

 - si no se especifican las dimensiones se inicializaran como 1
 
 -LIMITES DE TAMA�O DE BLOQUE
	- 1024 HILOS PARA DIMENSION X 
	- 1024 HILOS PARA DIMENSION Y
	-   64 HILOS PARA DIMENSION Z

	- x* y* x <= 1024	la multiplicacion del numero de subprocesos en cada 
	  dimension deber ser menor o igual a 1024

-LIMITES DE TAMA�O DE CUADRICULA
	- 65536 (1<<32-1)  BLOQUES PARA DIMENSION X
	- 65536 (2^32-1)  BLOQUES PARA DIMENSION Y,Z




*/


//// EJEMPLOS 
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
////kernel
///*
//* funcion asincrona ( el host puede continuar con las siguientes instrucciones) a
//* menos que se especifique que debe esperar (hipDeviceSynchronize())
//* 
//* hipDeviceReset();  reestablece el dispositivo
//*/
//__global__ void hello_cuda()
//{
//	printf("Hello CUDA world \n");
//	//cout << "hello CPU world" << endl; // esta instruccion no funciona dentro del kernel
//}
//
//
//int main() 
//{
//	//************************// EJEMPLO 1 ***
//	
//	//hello_cuda <<<1,1 >>>(); // kernel con parametros de lanzamiento
//	/*
//	* - El segundo parametro hace referencia al numero de subprocesos que se ejecutanran en el DEVICE
//	* 
//	*/
//	//hello_cuda << <1, 10 >> > (); // imprime 10 veces hello CUDA world
//
//	//hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE
//
//	//hipDeviceReset(); //reestablece dispositivo
//
//	//cout << "hello CPU world" << endl;
//
//	//return 0;
//
//	//************************// EJEMPLO 2 "8 BLOQUES(8X1) CON 4 HILOS(4X1)" imprime 32 veces hello CUDA world***
//
//	//dim3 grid(8); // conjunto de 8 blocks en X y 1 en las dimensiones Y,Z.
//	//dim3 block(4); // bloque con tama�o 4 en X y 1 en Y,Z.
//	//
//	//// el primer parametro(grid) es el numero de bloques de hilos en cada dimension
//	//// el segundo parametro(block) es el numero de hilos en cada dimension del bloque
//
//	//hello_cuda << <grid, block >> > (); 
//
//	//hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE
//
//	//hipDeviceReset(); //reestablece dispositivo
//
//	//cout << "hello CPU world" << endl;
//
//	//return 0;
//
//	//************************// EJEMPLO 3 "4 BLOQUES (2X2) CON 16 HILOS (8X2) " imprime 32 veces hello CUDA world***
//
//	
//	int nx; // variables dinamicas para ir modificando en tiempo de ejecucion
//	int ny;
//	nx = 16;
//	ny = 4;
//	
//	
//	dim3 block(8,2); // 16 hilos en cada bloque
//	dim3 grid(nx/block.x, ny/block.y); // 16/8=2 , 4/2=2  4 bloques en total
////		 _____________________________________________
////		|  | |   | |   | |   | |  |	|  | |   | |   | |
////		|  |1|	 |2|   |3|   |4|  |5|  |6|	 |7|   |8| == 4  BLOQUES (GRID) IGUALES A ESTE.
////		|  |1|	 |2|   |3|   |4|  |5|  |6|	 |7|   |8|
////		|__|_|___|_|___|_|___|_|__|_|__|_|___|_|___|_|
//	    
//	// el primer parametro(grid) es el numero de bloques de hilos en cada dimension
//	// el segundo parametro(block) es el numero de hilos en cada dimension del bloque
//
//	hello_cuda << <grid, block >> > ();
//
//	hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE
//
//	hipDeviceReset(); //reestablece dispositivo
//
//	cout << "hello CPU world" << endl;
//
//	return 0;
//
//}

//------------------------106 - Organization of threads in a CUDA program 1----------------------------------------------------

//1D
//      	 ______________________		     ____________________
//  		|  |A|   |B|   |C|   |D|		|E|  |F|     |G|   |H|

//Threadlx.X|  |0|	 |1|   |2|   |3|		|0|  |1|	 |2|   |3| 
//Threadlx.Y|  |0|	 |0|   |0|   |0|		|0|  |0|	 |0|   |0|  2 bloques : ejemplo de identificacion de hilo
//Threadlx.Z|  |0|	 |0|   |0|   |0|		|0|  |0|	 |0|   |0|				C = 2,0,0

//  		|__|_|___|_|___|_|___|_|		|_|__|_|_____|_|___|_|

//2D
//      	 __0_____1____2____3___		    _0___1_______2___3__
//  		|  ||   |X|   ||   ||			|P|  ||     ||   ||

//  		|  ||   |Y|   ||   ||			||   ||     |Q|  ||
//      	 ______________________		     ____________________
//																					X Y P Q			R S T U											
//      	 ______________________		     ____________________	 Threadlcx.X	1 1 0 2			0 3 1 0
//  		|  |R|   ||   ||   ||			||  |T|     ||   ||		 Threadlcx.Y	0 1 0 1			0 1 0 1	

//  		|  ||    ||   ||   |S|			|U|  ||     ||   ||
//      	 ______________________		     ____________________

//	//************************// EJEMPLO 1 -> GRID 2X2 CON 8 HILOS CADA BLOQUE ***
//	//************************// EJEMPLO 1 -> IDENTIFICACION DE HILOS ***

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//__global__ void print_threadIds()
//{
//	printf("threadIdx.x : %d,  threadIdx.y : %d,  threadIdx.z : %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
//}
//
//int main()
//{
//	int nx, ny;
//	nx = 2;
//	ny = 2;
//
//	dim3 block(2, 2); // 8 subprocesos en la dimension X y 8 subprocesos en la dimension Y.
//	dim3 grid(nx / block.x, ny / block.y);  // grid de 2x2
//
//	print_threadIds << <grid, block >> > ();
//	hipDeviceSynchronize(); // da la orden para que el host o int main espere a que termine el kernel o __global__.
//	hipDeviceReset();
//	return 0;
//
//	/*
//	ORDEN  GRID 2X2 = 4 BLOCKS , 4 HILOS POR BLOCK = 16 HILOS 
//
//	PRIMER BLOCK 1
//	  _____
//	 |	   |
//	 | A B |
//	 | C D |
//	 |_____|
//	
//	*BLOCKS					1	|	  2   |	   3	|    4
//						A B C D   E F G H   I J K L   M N O P			
//		Threadidx.X		0 1 0 1	  0 1 0 1	0 1 0 1	  0 1 0 1
//		Threadidx.Y		0 0 1 1	  0 0 1 1	0 0 1 1	  0 0 1 1
//		Threadidx.Z		0 0 0 0	  0 0 0 0	0 0 0 0	  0 0 0 0
//
//	*/
//}

//------------------------107 - Organization of threads in a CUDA program 2----------------------------------------------------

/*
* En tiempo de ejecucion CUDA la variable blckldx inicializada de forma unica para cada hilo dependiendo de las coordenadas de la pertenencia

blockldx.X = coordenadas de cada hilo tomando como base cada block

//1D				   0					        1
//      	 ____________________		    __________________
//  		  |P|   ||   |Q|   ||			||  |R|   ||   |S|   
//     0  	 ____________________		    __________________						P Q R S			T U V X
//																	blockldx.X		0 0	1 1			0 0 1 1
//      	 ____________________		    __________________		blockldx.Y		0 0 0 0			1 1 1 1
//  		  |T|   ||   |U|   ||			|V|  ||   ||   |X|
//     1  	 ____________________		    __________________

//--------------------------------------------------------------------------
//					__________________X_______________
//2D				   0					        1
//      	 ____________________		    ___________________
//  		  ||   |X|   ||   ||			|P|  ||     ||   ||
//    |0
//    |		  ||   |Y|   ||   ||			||   ||     |Q|  ||
//    |   	 ______________________		     ____________________
//	 Y|																				X Y P Q			R S T U
//    |		______________________		     ____________________	 blockldx.X		0 0 1 1			0 0 1 1
//    |		  |R|   ||   ||   ||			||  |T|     ||   ||		 blockldx.Y		0 0 0 0			1 1 1 1
//    |1															 blockDim.X = 4
//  		  ||    ||   ||   |S|			|U|  ||     ||   ||		 blockDim.Y = 2
//      	 ______________________		     ____________________	 GridDim.X =  2
																	 GridDim.Y =  2

blockDim = es la dimension del bloque ej. blockDim.x=4 y blockDim.y = 2 da como resultado un bloque de 8 hilos
GridDim = es la dimension de la rejilla ej. gridDim.x = 2 y gridDim.y = 2 da como resultado 4 bloques de hilos

*/
//EJEMPLO

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//__global__ void print_details()
//{
//	printf("blockIdx.x : %d,  blockIdx.y : %d,  blockIdx.z : %d, blockDim.x : %d,  blockDim.y : %d, gridDim.x : %d,  gridDim.y : %d \n", blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
//}
//
//int main()
//{
//	int nx, ny;
//	nx = 4;
//	ny = 1;
//
//	dim3 block(1, 1); // 8 subprocesos en la dimension X y 8 subprocesos en la dimension Y.
//	dim3 grid(nx / block.x, ny / block.y);  // grid de 2x2
//
//	print_details << <grid, block >> > ();
//	hipDeviceSynchronize(); // da la orden para que el host o int main espere a que termine el kernel o __global__.
//	hipDeviceReset();
//	return 0;
//}


//------------------------108 - Ejercicio grid 3d y block 3d----------------------------------------------------

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//__global__ void print_details()
//{
//	printf("threadIdx.x : %d,  threadIdx.y : %d,  threadIdx.z : %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
//	printf("blockIdx.x : %d,  blockIdx.y : %d,  blockIdx.z : %d, blockDim.x : %d,  blockDim.y : %d,  blockDim.z : %d, gridDim.x : %d,  gridDim.y : %d,  gridDim.z : %d \n", blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
//}
//
//int main()
//{
//	int nx, ny, nz;
//	nx = 4;
//	ny = 4;
//	nz = 4;
//
//	dim3 block(2, 2, 2); // 8 subprocesos en la dimension X y 8 subprocesos en la dimension Y.
//	dim3 grid(nx / block.x, ny / block.y, nz / block.z);  // grid de 2x2
//
//	print_details << <grid, block >> > ();
//	hipDeviceSynchronize(); // da la orden para que el host o int main espere a que termine el kernel o __global__.
//	hipDeviceReset();
//	return 0;
//}

//------------------------109 Unique index calculation using threadIdx blockId and blockDim--------------------
//************************************Ejemplo 1
////asignar valores de un array a cada hilo
//
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//__global__ void unique_idx_calc_threadIdx(int* input)
//{
//	int tid = threadIdx.x;
//	printf("threadIdx : %d, value : %d \n", tid, input[tid]);
//}
//
//int main()
//{
//	int array_size = 8;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23,9,4,53,65,12,1,33 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		cout << h_data[i] << " ";
//	}
//
//	cout << endl;
//	cout << endl;
//
//	int* d_data;
//	hipMalloc((void**)&d_data, array_byte_size);
//	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
//
//	//dim3 block(8); //8 threads en un bloque
//	//dim3 grid(1);
//
//	dim3 block(4); // 8 threads en 2 bloques de 4 cada uno
//	dim3 grid(2);
//
//	unique_idx_calc_threadIdx << <grid, block >> > (d_data);
//	hipDeviceReset();
//	return 0;
//
//
//}

//************************************//Ejemplo 2
//asignar valores de un array continuos a un grupo de blocks (grid 1D con 16 hilos en 4 bloques)
// gid = tid + offset
// gid = tid + blackldx.x * blockDim.x

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//
//
//__global__ void unique_gid_calculation(int * input)
//{
//	int tid = threadIdx.x;
//	int offset = blockIdx.x * blockDim.x; // numero de hilos que componen un bloque 
//	int gid = tid + offset; //indice en el que empezara a asignar valores a cada bloque de hilos
//
//	//ejemplo de 3 blocks
///*
//
////1D						   0					            1							      3
////			      	  _______________________		    ________________________		________________________
////  				      |23|   |9|   |4|   |53|			|65|   |12|   |1|   |33|		|65|   |12|   |1|   |33|
//tid(threadIdx)  =        0      1     2      3	          0     1      2      3			  0     1      2      3
//blockIdx.x	  =		   0	  0     0	   0			  1     1      1      1			  2     2      2      2
//blockDim.x      =        4	  4		4	   4			  4	    4	   4	  4			  4	    4	   4	  4
//offset		  =		   0      0     0      0              4     4      4      4			  8     8      8      8
//gid			  =		   0	  1     2      3			  4     5      6      7           8     9      10     11
//
//*/
//
//	printf("blockIdx.x : %d, threadIdx.x : %d, gid: %d, value : %d \n",
//		blockIdx.x, tid, gid, input[gid]);
//	 
//}
//
//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23,9,4,53,65,12,1,33,22,1,1,3,5,2,1,3 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		cout << h_data[i] << " ";
//	}
//
//	cout << endl;
//	cout << endl;
//
//	int* d_data;
//	hipMalloc((void**)&d_data, array_byte_size);
//	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
//
//	//dim3 block(8); //8 threads en un bloque
//	//dim3 grid(1);
//
//	dim3 block(4); // 4 threads en 4 bloques 
//	dim3 grid(4);
//
//	unique_gid_calculation << <grid, block >> > (d_data);
//	hipDeviceReset();
//	return 0;
//
//
//}


//------------------------110  Unique index calculation for 2D grid 1--------------------
//------------------------110  calculo del indice global para cuadricula 2D 1 (GRID DE 2X2 CON 4x1 hilos) ------
/*
* 
* Formula para calcular el indice unico para identificar los hilos que estan en una segunda fila
* 
* Index = row offset + block offset + tid
* row offset = number of threads in one thread block row (blockldx.y)
* block offset = number of threads in thread block(blockldx.x)
* tid = threadldx.x
* 
* gid = gridDim.x * blockDim.x * blockldx.y + blockldx.x * blockDim.x + threadldx.x

*/

//asignar valores de un array continuos a un grupo de blocks (grid 2D con 16 hilos en 4 bloques de 4x1 hilos)

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//
//
//__global__ void unique_gid_calculation_2d(int * input)
//{
//	int tid = threadIdx.x;
//	int offset = blockIdx.x * blockDim.x; // numero de hilos que componen un bloque 
//
//	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
//
//	int gid = tid + offset + row_offset; //indice en el que empezara a asignar valores a cada bloque de hilos

	//ejemplo de 3 blocks
/*
//2D (4 BLOQUES EN UN GRID DE 2X2
//								0					            1						
//			      	  _______________________		    ________________________		
//fila 1 de bloques   |23|   |9|    |4|   |53|			|22|   |1|   |1|   |3|		

//			      	  _______________________		    ________________________
//fila 2 de bloques	  |65|   |12|   |1|   |33|			|5|    |2|   |1|   |3|

//fila 1 de bloques   |23|   |9|    |4|   |53|			 |65|   |12|   |1|   |33|
tid(threadIdx.X)=      0      1     2      3	          0     1      2      3			 
blockIdx.x	  =		   0	  0     0	   0			  1     1      1      1			 
blockDim.x    =        4	  4		4	   4			  4	    4	   4	  4			 
offset		  =		   0      0     0      0              4     4      4      4			  
blockIdx.y    =		   0	  0		0	   0			  0		0	   0	  0
gridDim.x     =        2      2     2      2			  2		2	   2	  2
rowOffset	  =		   0	  0     0      0			  0     0      0      0          
gid			  =		   0	  1     2      3			  4     5      6      7          

//fila 2 de bloques	 |65|   |12|   |1|   |33|			 |5|    |2|   |1|    |3|
tid(threadIdx.X)=      0      1     2      3	          0     1      2      3
blockIdx.x	  =		   0	  0     0	   0			  1     1      1      1
blockDim.x    =        4	  4		4	   4			  4	    4	   4	  4
offset		  =		   0      0     0      0              4     4      4      4
blockIdx.y    =		   1	  1		1	   1			  1		1	   1	  1
gridDim.x     =        2      2     2      2			  2		2	   2	  2
rowOffset	  =		   8	  8     8      8			  8     8      8      8
gid			  =		   8	  9     10     11			 12    13     14     15
rowOffset = blockDim.x * gridDim.x * blockIdx.y;
gid = tid + offset + row_offset; //indice en el que empezara a asignar valores a cada bloque de hilos
*/

//	printf("blockIdx.x : %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d - input: %d \n",
//		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
//	 
//}
//
//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23,9,4,53,65,12,1,33,22,1,1,3,5,2,1,3 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		cout << h_data[i] << " ";
//	}
//
//	cout << endl;
//	cout << endl;
//
//	int* d_data;
//	hipMalloc((void**)&d_data, array_byte_size);
//	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
//
//	//dim3 block(8); //8 threads en un bloque
//	//dim3 grid(1);
//
//	dim3 block(4); // 4 threads en 4 bloques 
//	dim3 grid(2,2);
//
//	unique_gid_calculation_2d << <grid, block >> > (d_data);
//	hipDeviceReset();
//	return 0;
//
//
//}


//-----------------111  Unique index calculation for 2D grid 2--------------------
//-----------------111  calculo del indice global para cuadricula 2D  (GRID DE 2X2 CON 2x2 hilos) -----
/*
*
* Formula para calcular el indice unico para identificar los hilos que estan en una segunda fila
*
* Index = row offset + block offset + tid
* row offset = number of threads in one thread block row (blockldx.y)
* block offset = number of threads in thread block(blockldx.x)
* tid = threadldx.x
*
* gid = gridDim.x * blockDim.x * blockldx.y + blockldx.x * blockDim.x + threadldx.x

*/

//asignar valores de un array continuos a un grupo de blocks (grid 2D con 16 hilos en 4 bloques de 2x2 hilos)

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//using namespace std;
//
//
//
//__global__ void unique_gid_calculation_2d_2d(int * input)
//{
//	int tid = blockDim.x * threadIdx.y + threadIdx.x;
//	
//	int num_threads_in_a_block = blockDim.x * blockDim.y;
//	int block_offset = blockIdx.x * num_threads_in_a_block;
//
//	int num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
//	int row_offset = num_threads_in_a_row * blockIdx.y;
//
//	int gid = tid + block_offset + row_offset; //indice en el que empezara a asignar valores a cada bloque de hilos

	//ejemplo de 3 blocks
/*
//2D (4 BLOQUES EN UN GRID DE 2X2
//								   0					            1
//bloques 1 y 2    			___________					    __________
//fila 1					|23|   |9|    					|22|   |1|   
//fila 2					 |4|   |53|						|1|    |3|

//bloques 3 y 4				____________						__________
//fila 1					|65|   |12|   					|5|    |2|   
//fila 2					|1|    |33|						|1|    |3|

tid = blockDim.x * threadIdx.y + threadIdx.x;
num_threads_in_a_block = blockDim.x * blockDim.y;
block_offset = blockIdx.x * num_threads_in_a_block;
num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
row_offset = num_threads_in_a_row * blockIdx.y;
gid = tid + block_offset + row_offset;

//fila 1 de bloques			|23|   |9|    |4|   |53|		 |22|   |1|   |1|    |3|
blockDim.x    =				 2		2	   2     2			  2		2	   2      2
treadsIdx.x   =				 0		1	   0     1			  0	    1	   0	  1
treadsIdx.y   =				 0		0	   1     1			  0	    0	   1	  1
tid			  =				 0      1      2     3	          0     1      2      3
blockDim.y    =				 2		2	   2     2			  2		2	   2      2
num_threads_in_a_block =	 4      4      4     4	          4     4      4      4
blockIdx.x	  =				 0	    0      0     0			  1     1      1      1
block_offset  =				 0	    0      0     0			  4     4      4      4
gridDim.x     =				 2      2      2     2			  2		2	   2	  2
num_threads_in_a_row   =	 8      8      8     8			  8     8      8      8
blockIdx.y	  =				 0	    0      0     0			  0     0      0      0
rowOffset	  =				 0		0      0     0			  0     0      0      0
gid			  =				 0		1      2     3			  4     5      6      7

tid = blockDim.x * threadIdx.y + threadIdx.x;
num_threads_in_a_block = blockDim.x * blockDim.y;
block_offset = blockIdx.x * num_threads_in_a_block;
num_threads_in_a_row = num_threads_in_a_block * gridDim.x;
row_offset = num_threads_in_a_row * blockIdx.y;
gid = tid + block_offset + row_offset;

//fila 2 de bloques			|65|   |12|   |1|   |33|		 |5|    |2|   |1|    |3|
blockDim.x    =				 2		2	   2     2			  2		2	   2      2
treadsIdx.x   =				 0		1	   0     1			  0	    1	   0	  1
treadsIdx.y   =				 0		0	   1     1			  0	    0	   1	  1
tid			  =				 0      1      2     3	          0     1      2      3
blockDim.y    =				 2		2	   2     2			  2		2	   2      2
num_threads_in_a_block =	 4      4      4     4	          4     4      4      4
blockIdx.x	  =				 0	    0      0     0			  1     1      1      1
block_offset  =				 0	    0      0     0			  4     4      4      4
gridDim.x     =				 2      2      2     2			  2		2	   2	  2
num_threads_in_a_row   =	 8      8      8     8			  8     8      8      8
blockIdx.y	  =				 1	    1      1     1			  1	    1      1      1
rowOffset	  =				 8      8      8     8			  8     8      8      8
gid			  =				 8		9      10    11			  12    13     14     15

*/

//	printf("blockIdx.x : %d, blockIdx.y: %d, threadIdx.x: %d, gid: %d - input: %d \n",
//		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
//	 
//}
//
//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23,9,4,53,65,12,1,33,22,1,1,3,5,2,1,3 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		cout << h_data[i] << " ";
//	}
//
//	cout << endl;
//	cout << endl;
//
//	int* d_data;
//	hipMalloc((void**)&d_data, array_byte_size);
//	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);
//
//	//dim3 block(8); //8 threads en un bloque
//	//dim3 grid(1);
//
//	dim3 block(2,2); // 4 threads en cada block (2x2)
//	dim3 grid(2,2);  // 4 blocks (2x2)
//
//	unique_gid_calculation_2d_2d << <grid, block >> > (d_data);
//	hipDeviceReset();
//	return 0;
//
//
//}

//-----------------112	CUDA MEMORY TRANSFER --------------------------------------------------

/*
-Two devices
	-HOST ( cpu- memory) - CPU - CACHES AND DRAM
	-DEVICE  ( gpu - internal gpu memory)- SM (stream multiprocess) - CACHES AND DRAM

- Para transferir memoria entre el host y el dispositivo
	cudaMemCpy(
		destination ptr, source ptr,
				size in byte, direction)
				* ptr = puntero 
				*destination ptr = hostToDevice o DeviceToHost o HostToHost (cudamemcpyhtod, cudamemcpydtoh, cudamemcpydtod)
		
*/

//Ejemplo 1. pasar datos a memoria del device en un solo bloque de hilos


//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//#include <stdlib.h>
//#include <time.h>
//using namespace std;
//
//__global__ void mem_trs_test(int* input) // kernel que toma como un puntero a una matriz de enteros
//{
//	//cuadricula 1D con 2 bloques de hilos
//	int gid = blockIdx.x * blockDim.x + threadIdx.x; //indice global para acceder a elementos de la matriz
//	printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
//}
//
//
//
//int main()
//{
//	int size = 128; // tama�o de la matriz
//	int byte_size = size * sizeof(int);// cantidad de bytes que necesitamos para asignar a esta matriz 
//	int* h_input;	//asignar memoria del Host (la h_ es para indicar que es una variable del lenguaje principal)
//
//	//asignacion de memoria usando funcion malloc.
//	h_input = (int*)malloc(byte_size); // asinacion de bytes necesarios
//
//	//inicializacion aleatoria de la matriz con secuencia aleatoria de numeros
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < size; i++)
//	{
//		h_input[i] = (int)(rand() & 0xff);//valor aleatoria entre 0 y 255
//	}
//
//	int* d_input; // se utiliza d_ para indicar que es una variable de dispositivo
//
//	//asignacion de memoria en el dispositivo(gpu)
//	/*
//		C		   CUDA
//	malloc		hipMalloc -- asignar memoria
//	memset		hipMemset -- establece valores para una ubicacion de memoria dada
//	free		hipFree   -- recupera la ubicacion de memoria especificada
//	
//	*/
//
//	// ** = puntero doble o puntero a un puntero
//	// &d_input = especifica tama�o de la memoria
//	hipMalloc((void**)&d_input,byte_size); 
//
//	hipMemcpy(d_input,h_input,byte_size,hipMemcpyHostToDevice);// tranferir la matriz inicializada en el host al dispositivo
//	// h_input = puntero de origen
//	// d_input = puntero de destino en el device
//
//	//parametros de lanzamiento
//	dim3 block(64); // TODO: POR LO GENERAL SE MANTIENE EL TAMA�O EN MULTIPLOS DE 32
//	dim3 grid(2);
//
//	mem_trs_test << <grid, block >> > (d_input);
//	hipDeviceSynchronize();// hace que la ejecucion espere en este punto
//
//	hipFree(d_input); // recuperar memoria 
//	free(h_input); // recuperar memoria 
//
//	hipDeviceReset();
//	return 0;
//
//}

//***********************************************

//Ejemplo 2. pasar datos a memoria del device en varios bloques de hilos


//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <iostream>
//#include <stdlib.h>
//#include <time.h>
//using namespace std;
//
//__global__ void mem_trs_test2(int* input, int size) // kernel que toma como un puntero a una matriz de enteros
//{ //int size = tama�o matriz
//	
//	//cuadricula 1D con 2 bloques de hilos
//	int gid = blockIdx.x * blockDim.x + threadIdx.x; //indice global para acceder a elementos de la matriz
//	
//													 
//	// CON ESTA VERIFICACION SOLO SE UTILIZAN LOS HILOS QUE MANEJARAN DATOS DADO EL INPUT												 
//	/*if (gid < size)
//	{
//		printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
//	}*/
//
//	// SIN LA VERIFICACION SE ACCEDE A LOS HILOS DE TODO EL GRID AUN CUANDO NO MANEJEN DATOS
//	printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
//}
//
//
//
//int main()
//{
//	int size = 150; // tama�o de la matriz
//	int byte_size = size * sizeof(int);// cantidad de bytes que necesitamos para asignar a esta matriz 
//	int* h_input;	//asignar memoria del Host (la h_ es para indicar que es una variable del lenguaje principal)
//
//	//asignacion de memoria usando funcion malloc.
//	h_input = (int*)malloc(byte_size); // asinacion de bytes necesarios
//
//	//inicializacion aleatoria de la matriz con secuencia aleatoria de numeros
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < size; i++)
//	{
//		h_input[i] = (int)(rand() & 0xff);//valor aleatoria entre 0 y 255
//	}
//
//	int* d_input; // se utiliza d_ para indicar que es una variable de dispositivo

	//asignacion de memoria en el dispositivo(gpu)
	/*
		C		   CUDA
	malloc		hipMalloc -- asignar memoria
	memset		hipMemset -- establece valores para una ubicacion de memoria dada
	free		hipFree   -- recupera la ubicacion de memoria especificada

	*/

	// ** = puntero doble o puntero a un puntero
	// &d_input = especifica tama�o de la memoria
//	hipMalloc((void**)&d_input, byte_size);
//
//	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);// tranferir la matriz inicializada en el host al dispositivo
//	// h_input = puntero de origen
//	// d_input = puntero de destino en el device
//
//	//parametros de lanzamiento
//	dim3 block(32); // TODO: POR LO GENERAL SE MANTIENE EL TAMA�O EN MULTIPLOS DE 32
//	dim3 grid(5);
//
//	mem_trs_test2 << <grid, block >> > (d_input,size);
//	hipDeviceSynchronize();// hace que la ejecucion espere en este punto
//
//	hipFree(d_input); // recuperar memoria 
//	free(h_input); // recuperar memoria 
//
//	hipDeviceReset();
//	return 0;
//
//}

//-----------------112	exercise GRID 3D --------------------------------------------------

//-----------------114 Sum array example with validity check --------------------------------------------------
//#include "hip/hip_runtime.h"
//#include ""
////#include "cuda_common.cuh"
//
//#include <stdio.h>
//#include "common.h" // incluye metodo para comparar matrices
//
//// for random initialize
//#include <stdlib.h>
//#include <time.h>
//
//// for memset
//#include <cstring>
//using namespace std;
//
//__global__ void sum_array_gpu(int* a, int* b, int* c, int size)
//{
//	int gid = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (gid < size) // verificar si el indice global esta dentro del tama�o de nuestra matriz
//	{
//		c[gid] = a[gid] + b[gid];
//	}
//}
//
//// funcion para verificar resultado de gpu
//void sum_array_cpu(int* a, int* b, int* c, int size)
//{
//	for (int i = 0; i < size; i++)
//	{
//		c[i] = a[i] + b[i];
//	}
//}
//
//int main()
//{
//	int size = 10000; // tama�o de la matriz
//	int block_size = 128; // tama�o del bloque en 128
//	int num_bytes = size * sizeof(int); // tama�o necesario en bytes
//
//	// punteros host
//	int* h_a, * h_b, * gpu_results;
//	
//	int* h_c; // para verificacion en cpu
//
//	//asignacion de memoria para cada puntero
//	h_a = (int*)malloc(num_bytes);
//	h_b = (int*)malloc(num_bytes);
//	gpu_results = (int*)malloc(num_bytes);
//	
//	h_c = (int*)malloc(num_bytes);// para verificacion en cpu
//
//	//inicializacion aleatoria de cada matriz
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < size; i++)
//	{
//		h_a[i] = (int)(rand() & 0xFF); // valor generado entre 0 y 255
//	}
//	for (int i = 0; i < size; i++)
//	{
//		h_b[i] = (int)(rand() & 0xFF);
//	}
//
//	sum_array_cpu(h_a, h_b, h_c, size);
//
//	memset(gpu_results, 0, num_bytes);
//
//	// punteros device
//	int* d_a, * d_b, * d_c;
//	hipMalloc((int**)&d_a, num_bytes);
//	hipMalloc((int**)&d_b, num_bytes);
//	hipMalloc((int**)&d_c, num_bytes);
//
//	//tranferencia de matriz h_a y h_b
//	hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, h_b, num_bytes, hipMemcpyHostToDevice);
//
//	//launching the grid
//	dim3 block(block_size); //tama�o de bloque 128 en la dimension X
//	dim3 grid((size / block.x) + 1); // (10000 / 128) + 128 = GRID 1D de 79 block de 128 hilos cada uno
//
//	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);
//	hipDeviceSynchronize();
//
//	hipMemcpy(gpu_results, d_c, num_bytes, hipMemcpyDeviceToHost); // puntero de origen d_c, puntero de destino gpu_results
//
//	// COMPARACION DE RESULTADOS CPU Y GPU
//	compare_arrays(gpu_results, h_c, size);
//	
//	hipFree(d_c);
//	hipFree(d_b);
//	hipFree(d_a);
//	 
//	free(gpu_results);
//	free(h_b);
//	free(h_a);
//
//	hipDeviceReset();
//	return 0;
//
//
//}

//-----------------116 Error handling --------------------------------------------------

/*Types error
*	-Compile time errors
*		-Errors language syntax.
* 
*	-Run time errors
*		-Errors happens while program is running
* 
*/

//ejemplo  con la suma anterior

#include "hip/hip_runtime.h"
#include ""
//#include "cuda_common.cuh"

#include <stdio.h>
#include "common.h" // incluye metodo para comparar matrices

// for random initialize
#include <stdlib.h>
#include <time.h>

// for memset
#include <cstring>

#include "cuda_common.cuh"

using namespace std;

__global__ void sum_array_gpu(int* a, int* b, int* c, int size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size) // verificar si el indice global esta dentro del tama�o de nuestra matriz
	{
		c[gid] = a[gid] + b[gid];
	}
}

// funcion para verificar resultado de gpu
void sum_array_cpu(int* a, int* b, int* c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int size = 10000; // tama�o de la matriz
	int block_size = 128; // tama�o del bloque en 128
	int num_bytes = size * sizeof(int); // tama�o necesario en bytes

	//ERROR (comprobacion)
	hipError_t error;

	// punteros host
	int* h_a, * h_b, * gpu_results;
	
	int* h_c; // para verificacion en cpu

	//asignacion de memoria para cada puntero
	h_a = (int*)malloc(num_bytes);
	h_b = (int*)malloc(num_bytes);
	gpu_results = (int*)malloc(num_bytes);
	
	h_c = (int*)malloc(num_bytes);// para verificacion en cpu

	//inicializacion aleatoria de cada matriz
	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++)
	{
		h_a[i] = (int)(rand() & 0xFF); // valor generado entre 0 y 255
	}
	for (int i = 0; i < size; i++)
	{
		h_b[i] = (int)(rand() & 0xFF);
	}

	sum_array_cpu(h_a, h_b, h_c, size);

	memset(gpu_results, 0, num_bytes);

	// punteros device
	int* d_a, * d_b, * d_c;
	
	//---------------------------
	//ERROR FORMA MANUAL
	/*error = hipMalloc((int**)&d_a, num_bytes);
	if (error != hipSuccess)
	{
		fprintf(stderr, "Error : %s \n", hipGetErrorString(error));
	}*/

	//ERROR UTILIZANDO cuda_common.cuh
	
	gpuErrchk(hipMalloc((int**)&d_a, num_bytes));
	gpuErrchk(hipMalloc((int**)&d_b, num_bytes));
	gpuErrchk(hipMalloc((int**)&d_c, num_bytes));

	//-------------------------------
	//hipMalloc((int**)&d_a, num_bytes);
	//hipMalloc((int**)&d_b, num_bytes);
	//hipMalloc((int**)&d_c, num_bytes);

	//tranferencia de matriz h_a y h_b
	hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, num_bytes, hipMemcpyHostToDevice);

	//launching the grid
	dim3 block(block_size); //tama�o de bloque 128 en la dimension X
	dim3 grid((size / block.x) + 1); // (10000 / 128) + 128 = GRID 1D de 79 block de 128 hilos cada uno

	sum_array_gpu << <grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();

	hipMemcpy(gpu_results, d_c, num_bytes, hipMemcpyDeviceToHost); // puntero de origen d_c, puntero de destino gpu_results

	// COMPARACION DE RESULTADOS CPU Y GPU
	compare_arrays(gpu_results, h_c, size);
	
	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	 
	free(gpu_results);
	free(h_b);
	free(h_a);

	hipDeviceReset();
	return 0;


}