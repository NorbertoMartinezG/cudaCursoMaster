
#include "hip/hip_runtime.h"


#include <stdio.h>

/*
02- Introducction to parallel programing

Context (instruccion por turnos)
	- Collection of data about process which allows processor to suspend or hold the execution of a process and restart the execution later.
	- Memory addresses
	- Program counter states

Thread (secuendia mas peque�a de instruccion programada
	- Process
		- threads (subprocess)

Parallel Process
	- Tipos
		- Paralelismo a nivel de tarea
			- los nucleos realizan tareas distintas con datos distintos o los mismos
		- Paralelismo a nivel de datos
			- los nucleos realizan la misma tarea con diferentes datos

Paralelismo vs concurrencia.
	- Concurrencia = realizacion de procesos en distintos tiempos(secuenciales) de milesimas de segundo que aparentan simultaneidad o paralelismo
	- Paralelismo = distintos nucleos realizan tareas al mismo tiempo

*/