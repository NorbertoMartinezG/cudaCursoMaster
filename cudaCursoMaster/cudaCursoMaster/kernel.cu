#include "hip/hip_runtime.h"

//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>

/*
------------------------02- Introducction to parallel programing -------------------------------------------------------------------------------------------------------------------------------------------------------

Context (instruccion por turnos)
	- Collection of data about process which allows processor to suspend or hold the execution of a process and restart the execution later.
	- Memory addresses
	- Program counter states

Thread (secuendia mas peque�a de instruccion programada
	- Process
		- threads (subprocess)

Parallel Process
	- Tipos
		- Paralelismo a nivel de tarea
			- los nucleos realizan tareas distintas con datos distintos o los mismos
		- Paralelismo a nivel de datos
			- los nucleos realizan la misma tarea con diferentes datos

Paralelismo vs concurrencia.
	- Concurrencia = realizacion de procesos en distintos tiempos(secuenciales) de milesimas de segundo que aparentan simultaneidad o paralelismo
	- Paralelismo = distintos nucleos realizan tareas al mismo tiempo

------------------------04- Install -------------------------------------------------------------------------------------------------------------------------------------------------------
Revisar compatibilidad en wikipedia en ingles.
GPGPU- 
windows + r (dxdiag) // visualiza las caracteristicas del PC 
windows + r (cmd) //escribir (nvcc --version) para saber la version de CUDA instalado del PC

*/

//------------------------005 - Basic steps of a CUDA program------------------
/*RESUMEN
* - initization of data from CPU
* - transfer data from CPU context to GPU context
* - Kernel launc with needed grid/block size
* - Transfer results back to CPU context from CPU context
* - Reclaim the memory from both CPU and GPU
* 
* - IMPORTANTE
*	-Grid - Grid is a collection of all the threads launch for a kernel ( coleccion de todos los hilos lanzados para un kernel)
*		- En el ejemplo hello CUDA world se tienen 20 subprocesos, los hilos en una cuadricula estan organizados
*		  en un grupo llamado bloques de hilos
*	-Block - subconjunto de hilos dentro de un GRID que se pueden representar como un cubo(3d) mas peque�o que a su vez esta subdividido en peque�os cubos que representan a los hilos o threads
	
	-GRID (cubo general en x, y z)
		-BLOCK (subcubo dentro de GRID que forma un subconjunto de hilos)
			-THREADS

	kernel_ name <<<
				number_of_blocks, // especifica cuantos bloques de hilos en la cuadracula en cada dimension
				thread_per_block  // especifica cuantos hilos en un bloque en cada dimension
								>>> (arguments) // TODO ESTO ES EN UNA DIMENSION

* - Para especificar cuadriculas y bloques multidimensionales
*	-dim3 variable_name (x,y,z) // se inicializa por defecto en 1
*		- dim3 variable_name(x,y,z) // puede acceder a cada valor de dimension 
*			- variable_name.x
* *			- variable_name.y
* *			- variable_name.z

*EJEMPLO UNIDIMENSIONAL
* - 8 bloques de hilos, donde cada bloque tiene 4 hilos en la dimension x
	-la dimension de nuestro bloque es de cuatro hilos en la dimension x y 1 hilo en las dimensiones Y y Z.
		
		-dim3 grid(8,1,1) // nos referimos a todos los hilos lanzados para un kernel como grid.
		-dim3 block(4,1,1)
		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________		 _________________________
		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |		|  | |   | |   | |   | |  |
		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |		|  |1|	 |2|   |3|   |4|  |	
		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|		|__|_|___|_|___|_|___|_|__|

 8 bloques unidimensionales con 8 hilos unidimensionales

 - si no se especifican las dimensiones se inicializaran como 1
 
 -LIMITES DE TAMA�O DE BLOQUE
	- 1024 HILOS PARA DIMENSION X 
	- 1024 HILOS PARA DIMENSION Y
	-   64 HILOS PARA DIMENSION Z

	- x* y* x <= 1024	la multiplicacion del numero de subprocesos en cada 
	  dimension deber ser menor o igual a 1024

-LIMITES DE TAMA�O DE CUADRICULA
	- 65536 (1<<32-1)  BLOQUES PARA DIMENSION X
	- 65536 (2^32-1)  BLOQUES PARA DIMENSION Y,Z




*/


// EJEMPLOS 
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
using namespace std;

//kernel
/*
* funcion asincrona ( el host puede continuar con las siguientes instrucciones) a
* menos que se especifique que debe esperar (hipDeviceSynchronize())
* 
* hipDeviceReset();  reestablece el dispositivo
*/
__global__ void hello_cuda()
{
	printf("Hello CUDA world \n");
	//cout << "hello CPU world" << endl; // esta instruccion no funciona dentro del kernel
}


int main() 
{
	//************************// EJEMPLO 1 ***
	
	//hello_cuda <<<1,1 >>>(); // kernel con parametros de lanzamiento
	/*
	* - El segundo parametro hace referencia al numero de subprocesos que se ejecutanran en el DEVICE
	* 
	*/
	//hello_cuda << <1, 10 >> > (); // imprime 10 veces hello CUDA world

	//hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE

	//hipDeviceReset(); //reestablece dispositivo

	//cout << "hello CPU world" << endl;

	//return 0;

	//************************// EJEMPLO 2 "8 BLOQUES(8X1) CON 4 HILOS(4X1)" imprime 32 veces hello CUDA world***

	//dim3 grid(8); // conjunto de 8 blocks en X y 1 en las dimensiones Y,Z.
	//dim3 block(4); // bloque con tama�o 4 en X y 1 en Y,Z.
	//
	//// el primer parametro(grid) es el numero de bloques de hilos en cada dimension
	//// el segundo parametro(block) es el numero de hilos en cada dimension del bloque

	//hello_cuda << <grid, block >> > (); 

	//hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE

	//hipDeviceReset(); //reestablece dispositivo

	//cout << "hello CPU world" << endl;

	//return 0;

	//************************// EJEMPLO 3 "4 BLOQUES (2X2) CON 16 HILOS (8X2) " imprime 32 veces hello CUDA world***

	
	int nx; // variables dinamicas para ir modificando en tiempo de ejecucion
	int ny;
	nx = 16;
	ny = 4;
	
	
	dim3 block(8,2); // 16 hilos en cada bloque
	dim3 grid(nx/block.x, ny/block.y); // 16/8=2 , 4/2=2  4 bloques en total
//		 _____________________________________________
//		|  | |   | |   | |   | |  |	|  | |   | |   | |
//		|  |1|	 |2|   |3|   |4|  |5|  |6|	 |7|   |8| == 4  BLOQUES (GRID) IGUALES A ESTE.
//		|  |1|	 |2|   |3|   |4|  |5|  |6|	 |7|   |8|
//		|__|_|___|_|___|_|___|_|__|_|__|_|___|_|___|_|
	    
	// el primer parametro(grid) es el numero de bloques de hilos en cada dimension
	// el segundo parametro(block) es el numero de hilos en cada dimension del bloque

	hello_cuda << <grid, block >> > ();

	hipDeviceSynchronize(); // hace que el CPU o host espere en este punto, hasta que termine el proceso de DEVICE

	hipDeviceReset(); //reestablece dispositivo

	cout << "hello CPU world" << endl;

	return 0;

}

